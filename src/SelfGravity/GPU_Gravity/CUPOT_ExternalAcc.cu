#ifdef __HIPCC__
#include "Macro.h"
#else
#include "GAMER.h"
#endif
#include "CUPOT.h"

#ifdef GRAVITY

// soften length implementation
//#  define SOFTEN_PLUMMER
//#  define SOFTEN_RUFFERT
#   define DRIV_TURB

#  if   (defined DRIV_TURB)
extern double* ExtAcc_InitialField[3];
#  endif




//-----------------------------------------------------------------------------------------
// Function    :  CUPOT_ExternalAcc / CPU_ExternlAcc
// Description :  Calculate the external acceleration at the given coordinates and time
//
// Note        :  1. This function will be invoked by both CPU and GPU
//                2. "__forceinline__" is required since this device function will be invoked
//                   by more than one kernels (e.g., CUPOT_HydroGravitySolver, CUFLU_ComputeFlux)
//                3. The auxiliary array "UserArray" is set by "Init_ExternalAcc_Ptr", which
//                   points to "Init_ExternalAcc()" by default but may be overwritten by various
//                   test problem initializers
//                4. By default we assume
//                     UserArray[0] = x coordinate of the external acceleration center
//                     UserArray[1] = y ...
//                     UserArray[2] = z ..
//                     UserArray[3] = gravitational_constant*point_source_mass
//                     UserArray[4] = soften_length (<=0.0 --> disable)
//                   --> but one can easily modify this file to change the default behavior
//                5. Two different soften length implementations are supported
//                   --> SOFTEN_PLUMMER & SOFTEN_RUFFERT
//
// Parameter   :  Acc       : Array to store the output external acceleration
//                x/y/z     : Target spatial coordinates
//                Time      : Current physical time
//                UserArray : User-provided auxiliary array (set by "Init_ExternalAcc_Ptr")
//
// Return      :  Acc
//-----------------------------------------------------------------------------------------
#ifdef __HIPCC__
__forceinline__ __device__
void CUPOT_ExternalAcc( real Acc[], const double x, const double y, const double z, const double Time, const double UserArray[] )
#else
void   CPU_ExternalAcc( real Acc[], const double x, const double y, const double z, const double Time, const double UserArray[] )
#endif
{
   const double Cen[3] = { UserArray[0], UserArray[1], UserArray[2] };
   const real GM       = (real)UserArray[3];
   const real eps      = (real)UserArray[4];
   const real dx       = (real)(x - Cen[0]);
   const real dy       = (real)(y - Cen[1]);
   const real dz       = (real)(z - Cen[2]);
   const real r        = SQRT( dx*dx + dy*dy + dz*dz );

// Plummer
#  if   ( defined SOFTEN_PLUMMER )
   const real _r3 = ( eps <= (real)0.0 ) ? (real)1.0/CUBE(r) : POW( SQR(r)+SQR(eps), (real)-1.5 );

// Ruffert 1994
#  elif ( defined SOFTEN_RUFFERT )
   const real tmp = EXP( -SQR(r)/SQR(eps) );
   const real _r3 = ( eps <= (real)0.0 ) ? (real)1.0/CUBE(r) : POW( SQR(r)+SQR(eps)*tmp, (real)-1.5 )*( (real)1.0 - tmp );

#  else
   const real _r3 = (real)1.0/CUBE(r);
#  endif

   Acc[0] = -GM*_r3*dx;
   Acc[1] = -GM*_r3*dy;
   Acc[2] = -GM*_r3*dz;

#  if   (defined DRIV_TURB)
   int m_temp, ix, iy, iz;
  
   ix = (int) ((x - amr->BoxEdgeL[0])/(amr->BoxEdgeR[0] - amr->BoxEdgeL[0]) * 256);
   iy = (int) ((y - amr->BoxEdgeL[1])/(amr->BoxEdgeR[1] - amr->BoxEdgeL[1]) * 256);
   iz = (int) ((z - amr->BoxEdgeL[2])/(amr->BoxEdgeR[2] - amr->BoxEdgeL[2]) * 256);

   if (ix < 0) ix += 256;
   if (iy < 0) iy += 256;
   if (iy < 0) iz += 256;
   if (ix > 255) ix -= 256;
   if (iy > 255) iy -= 256;
   if (iz > 255) iz -= 256;

   if((ix < 0 || ix > 255) || (iy < 0 || iy > 255) || (iz < 0 || iz > 255))
     Aux_Message(stderr, "At %lf %lf %lf index %d %d %d\n",
        x, y, z, ix, iy, iz);

   m_temp = (iz + 256 * (iy + 256 * ix));

   Acc[0] = ExtAcc_InitialField[0][m_temp];
   Acc[1] = ExtAcc_InitialField[1][m_temp];
   Acc[2] = ExtAcc_InitialField[2][m_temp];
  
   //free( m );
   //if ((ix == iy) && (iy == iz) && (iz == 0)) 
   // Aux_Message(stderr, "At %lf %lf %lf acc %lf %lf %lf\n",
   //     x, y, z, Acc[0], Acc[1], Acc[2]);

#  endif

} // FUNCTION : CUPOT_ExternalAcc / CPU_ExternalAcc



#endif // #ifdef GRAVITY
